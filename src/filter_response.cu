#include "hip/hip_runtime.h"
#include "filter_response.h"
//#include "logging.h"


//---------------------------------------------------------

__global__ void kernel_fconv(FPTYPE *gpuA, int A_dim0, int A_dim1, int A_dim2, FPTYPE *gpuB, int B_dim0, int B_dim1, int B_dim2, FPTYPE *gpuCi, int C_dim0, int C_dim1, int num_features)
{
	int f = blockIdx.z*blockDim.z + threadIdx.z;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x >= C_dim1 || y >= C_dim0 || f >= num_features )
		return; // skip padding thread

	FPTYPE *dst = gpuCi + f*C_dim0*C_dim1 ;
	FPTYPE *A_src = gpuA + f*A_dim0*A_dim1;      
	FPTYPE *B_src = gpuB + f*B_dim0*B_dim1;
	FPTYPE val = 0;
	for(int xp = 0; xp < B_dim1; xp++) 
	{
		FPTYPE *A_off = A_src + (x+xp)*A_dim0 + y;
		FPTYPE *B_off = B_src + xp*B_dim0;
		for(int yp = 0; yp < B_dim0; yp++)
			val += *(A_off++) * *(B_off++);
	}
	dst[x*C_dim0 + y] = val;
}

__global__ void kernel_sum_Ci(FPTYPE *gpuCi, int C_dim0, int C_dim1, int num_features)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x >= C_dim1 || y >= C_dim0 )
		return; // skip padding thread

	FPTYPE val = 0;
	for(int f = 0; f < num_features; f++)
		val += gpuCi[f*C_dim0*C_dim1 + x*C_dim0 + y];

	gpuCi[x*C_dim0 + y] = val;
}

//---------------------------------------------------------

#define BKSIZE 4
const int STREAMSNBR = 2;

// divide x by y and round up result
#define divup(x,y)  1+(((x)-1)/(y))  // if x != 0 only

hipStream_t stream[STREAMSNBR];

void initCuda(void)
{
	for(int i = 0; i < STREAMSNBR; i++)
    	hipStreamCreate(&stream[i]);
}

void releaseCuda(void)
{
	for(int i = 0; i < STREAMSNBR; i++)
		hipStreamDestroy(stream[i]);
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();
}

myArray<FPTYPE>** fconv_cuda(const myArray<FPTYPE> *pyra_feat_level, myArray<FPTYPE> **filters, int start, int end)
{
	// get A (features)
	const myArray<FPTYPE> *mxA = pyra_feat_level;
	FPTYPE *A = mxA->getPtr();
	int num_features = mxA->getPlanes();
	int A_dim0 = mxA->getRows();
	int A_dim1 = mxA->getCols();
	int A_dim2 = mxA->getPlanes();

	// get B (filters) and start/end
	myArray<FPTYPE> **cellB = filters;
	start = start - 1;
	end = end - 1;
	int len = end - start + 1;

	// allocate output
	myArray<FPTYPE> **C = (myArray<FPTYPE>**) calloc(len, sizeof(myArray<FPTYPE>*));

	// use 2 streams to overlapp memory transfer and computation
	int currentStream = 0;
	int nextStream = 1;

	// copy A on GPU
	int A_bytes = mxA->getNumberOfElements() * sizeof(FPTYPE);
	FPTYPE *gpuA;
	hipMalloc((void**) &gpuA, A_bytes);
	hipMemcpyAsync(gpuA, A, A_bytes, hipMemcpyHostToDevice, stream[currentStream]);

	// memory for B anc Ci on GPU is allocated only at the first iteration
	// because B and Ci size decrease

	// allocate memory for output C[i] on GPU (1 C[i] per feature)
	// C[i] is smaller than A, so we use the size of A to over-allocate
	// memory for C
	// allocate gpu memory just the first time, and not at each iteration
	FPTYPE *gpuCi[STREAMSNBR] = { NULL };
	int Ci_bytes_max = A_dim0 * A_dim1 * num_features * sizeof(FPTYPE);
	for(int i = 0; i < STREAMSNBR; i++)
	{
		hipMalloc((void**) &gpuCi[i], Ci_bytes_max);
		hipMemset((void*) gpuCi[i], 0, Ci_bytes_max);
	}

	// pre-load data on GPU for first iteration
	FPTYPE *gpuB[STREAMSNBR] = { NULL };
	const myArray<FPTYPE> *mxB = cellB[0+start];
	FPTYPE *B = mxB->getPtr();
	int B_dim0 = mxB->getRows();
	int B_dim1 = mxB->getCols();
	int B_dim2 = mxB->getPlanes();

	// copy first B array on GPU
	// allocate gpu memory just the first time, and not at each iteration
	int B_bytes_first = mxB->getNumberOfElements() * sizeof(FPTYPE);
	for(int i = 0; i < STREAMSNBR; i++)
		hipMalloc((void**) &gpuB[i], B_bytes_first);
	hipMemcpyAsync(gpuB[currentStream], B, B_bytes_first, hipMemcpyHostToDevice, stream[currentStream]);

	// loop over parts and filters
	for (int i = 0; i < len; i++)
	{
		if( A_dim2 != B_dim2 )
			throw std::runtime_error("fconv_cuda(): invalid input B");

		nextStream = (currentStream+1) % STREAMSNBR;

		// pre-load B on GPU for next iteration
		if( i < len-1 )
		{
			mxB = cellB[i+1+start];
			B = mxB->getPtr();
			int B_bytes = mxB->getNumberOfElements() * sizeof(FPTYPE);
			hipMemcpyAsync(gpuB[nextStream], B, B_bytes, hipMemcpyHostToDevice, stream[nextStream]);
		}

		// compute size of output
		int height = A_dim0 - B_dim0 + 1;
		int width = A_dim1 - B_dim1 + 1;
		if( height < 1 || width < 1 )
			throw std::runtime_error("fconv_cuda(): invalid input: B should be smaller than A");
		int C_dim0 = height;
		int C_dim1 = width;

		// compute C[i] for all features
		dim3 dimBlock(BKSIZE, BKSIZE, 32);
		dim3 dimGrid(divup(width, dimBlock.x), divup(height, dimBlock.y), divup(num_features,dimBlock.z));
		//printf("w, h, f = %d %d %d\n", width, height, num_features);
		//printf("dimBlock = %d %d %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
		//printf("dimGrid = %d %d %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
		kernel_fconv<<< dimGrid, dimBlock, 0, stream[currentStream] >>>(gpuA, A_dim0, A_dim1, A_dim2, gpuB[currentStream], B_dim0, B_dim1, B_dim2, gpuCi[currentStream], C_dim0, C_dim1, num_features);

		//DBG
		/*
		myArray<FPTYPE> *tmpCis = new myArray<FPTYPE>(A_dim0, A_dim1, num_features);
		FPTYPE *tmpCisPtr = tmpCis->getPtr();
		hipMemcpyAsync(tmpCisPtr, gpuCi, Ci_bytes_max, hipMemcpyDeviceToHost, stream[currentStream]);
		writeLog("fconv_cuda tmpCis=");
		writeLog(tmpCis);
		delete tmpCis;
		*/

		// sum features contributions
		dim3 dimBlock2(BKSIZE, BKSIZE);
		dim3 dimGrid2(divup(width, dimBlock.y), divup(height, dimBlock.x));
		kernel_sum_Ci<<< dimGrid2, dimBlock2, 0, stream[currentStream] >>>(gpuCi[currentStream], C_dim0, C_dim1, num_features);

		// transfer result from GPU memory to CPU memory
		C[i] = new myArray<FPTYPE>(height, width, 1, true);
		FPTYPE *Ci = C[i]->getPtr();
		int Ci_bytes = C[i]->getNumberOfElements() * sizeof(FPTYPE);
		hipMemcpyAsync(Ci, gpuCi[currentStream], Ci_bytes, hipMemcpyDeviceToHost, stream[currentStream]);

		// prepare next iteration
		currentStream = nextStream;
		B_dim0 = mxB->getRows();
		B_dim1 = mxB->getCols();
		B_dim2 = mxB->getPlanes();
	}

	for(int i = 0; i < STREAMSNBR; i++)
		hipStreamSynchronize(stream[i]);

	// release GPU memory
	for(int i = 0; i < STREAMSNBR; i++)
	{
		hipFree(gpuCi[i]);
		hipFree(gpuB[i]);
	}
	hipFree(gpuA);

	return C;
}

void cudaWakeUp(void)
{
	// make a dummy memory allocation to wake up NVIDIA driver
	// before starting time measurement, in order to make fair comparison
	int *gpuWakeUp;
	hipMalloc((void**) &gpuWakeUp, 1);
	hipFree(gpuWakeUp);
}
